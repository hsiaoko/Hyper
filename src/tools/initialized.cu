#include "hip/hip_runtime.h"
#include <stdio.h>

//CUDA 初始化
bool InitCUDA(){
    int count;
    hipGetDeviceCount(&count);//取得支持Cuda的装置的数目
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }
    int i;
    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }
    hipSetDevice(i);
    return true;
}
/*
int main(){ 
    if (!InitCUDA()) {
        return 0;
    }//CUDA 初始化
    printf("CUDA initialized.\n");
    return 0;
}
*/
