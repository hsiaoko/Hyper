#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %d.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %d.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);   // 输出的是GPU的时钟频率
    printf("textureAlignment : %d.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

bool InitCUDA()
{
    //used to count the device numbers
    int count;

    // 获取CUDA设备数 
    hipGetDeviceCount(&count);  
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    // find the device >= 1.X
    int i;
    for (i = 0; i < count; ++i) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {  //hipGetDeviceProperties获取 CUDA 设备属性
            if (prop.major >= 1) {
                printDeviceProp(prop);
                break;
            }
        }
    }

    // if can't find the device
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    // set cuda device
    hipSetDevice(i);

    return true;
}
/*
int main(int argc, char const *argv[])
{
    if (InitCUDA()) {
        printf("CUDA initialized.\n");
    }

    return 0;
}
*/